/*
 Copyright (c) 2021 Temporal Guild Group, Austral University of Chile, Valdivia Chile.
 This file and all powermon software is licensed under the MIT License. 
 Please refer to LICENSE for more details.
 */

#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <map>
#include <random>
#include <cmath>
#include <omp.h>
#define REAL float
#define TCSIZE 16
#define TCSQ 256
#define PRINTLIMIT 2560
#define WARPSIZE 32
#define DIFF (BSIZE<<3)

#define OMP_REDUCTION_FLOAT "omp-nt" STR(NPROC) "-float"
#define OMP_REDUCTION_DOUBLE "omp-nt" STR(NPROC) "-double"

#define STR_HELPER(x) #x
#define STR(x) STR_HELPER(x)

#include "nvmlPower.hpp"


int main(int argc, char **argv){
    if(argc != 2){
        fprintf(stderr, "\nrun as ./powermon dt-ms\ndt-ms: sample interval in milliseconds\n\n");
        exit(EXIT_FAILURE);
    }
    int ms = atoi(argv[1]); 
    // begin
    printf("Press enter to finalize...\n");
    GPUPowerBegin("gpu", ms);
    CPUPowerBegin("cpu", ms);

    printf("enter para terminar\n"); fflush(stdout);
    getchar();
    // end
    GPUPowerEnd();
    CPUPowerEnd();
    exit(EXIT_SUCCESS);
}
